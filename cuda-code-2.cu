
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *a) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] = threadIdx.x;
}

int main() {
    int dimx = 16;
    int num_bytes = dimx * sizeof (int);

    // device and host pointers
    int *d_a = 0; 
    int *h_a = 0; 

    h_a = (int*) malloc(num_bytes);
    hipMalloc((void**) &d_a, num_bytes);

    if (0 == h_a || 0 == d_a) {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset(d_a, 0, num_bytes);

    dim3 grid, block;
    block.x = 4;
    grid.x = dimx / block.x;

    kernel <<<grid, block>>>(d_a);

    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < dimx; i++)
        printf("%d ", h_a[i]);
    printf("\n");

    free(h_a);
    hipFree(d_a);

    return 0;
}
