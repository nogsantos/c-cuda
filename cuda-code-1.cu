
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int dimx = 10;
    int num_bytes = dimx * sizeof (int);
    
    // device and host pointers
    int *d_a = 0;
    int *h_a = 0; 
    /*Aloca mem�ria na CPU para n inteiros*/
    h_a = (int*) malloc(num_bytes);
    printf("%i\n", num_bytes);
    /*Aloca mem�ria na GPU para n inteiros*/
    hipMalloc(&d_a, num_bytes);

    if (0 == h_a || 0 == d_a) {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset(d_a, 0, num_bytes);
    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

    
    for (int i = 0; i < dimx; i++){
        printf("%d ", i+1);
    }
    printf("\n");

    free(h_a);
    hipFree(d_a);
    
    return 0;
    
}