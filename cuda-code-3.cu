
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *a, int dimx, int dimy, int n) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * dimx + ix;

    a[idx] = n;
}

int main() {
    int dimx = 16;
    int dimy = 16;
    int num_bytes = dimx * dimy * sizeof (int);

    int *d_a = 0, *h_a = 0; // device and host pointers

    h_a = (int*) malloc(num_bytes);
    hipMalloc((void**) &d_a, num_bytes);

    if (0 == h_a || 0 == d_a) {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset(d_a, 0, num_bytes);

    dim3 grid, block;
    block.x = 4;
    block.y = 4;
    grid.x = dimx / block.x;
    grid.y = dimy / block.y;

    kernel <<<grid, block>>>(d_a, dimx, dimy, 2);

    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

    for (int row = 0; row < dimy; row++) {
        for (int col = 0; col < dimx; col++){
            printf("%d ", h_a[row * dimx + col]);
        }
        printf("\n");
    }

    free(h_a);
    hipFree(d_a);

    return 0;
}
